#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

// Each thread will calculate the dot product of a row of the matrix and the vector
__global__ void spmv(int * row_ptrs, int * col_offsets, int * data, int * vector, int * result, int vector_size) {
    int i = threadIdx.x;
    int start = row_ptrs[i], end = row_ptrs[i + 1];
    int sum = 0;
    for (int j = start; j < end; j++) {
        sum += data[j] * vector[col_offsets[j]];
    }
    result[i] = sum;
}

void to_csr_format(int * mat, int r, int c, int ** row_ptrs, int ** col_offsets, int ** data, int * ret_row_ptrs_count, int * ret_data_count) {
    int row_ptrs_count = 0, data_count = 0;
    for (int i = 0; i < r; i++) {
        int inserted_row_ptr = 0;
        for (int j = 0; j < c; j++) {
            int ele_ind = i * c + j;
            if (mat[ele_ind] != 0) {
                if (inserted_row_ptr == 0) {
                    row_ptrs_count++;
                    *row_ptrs = (int *) realloc(*row_ptrs, sizeof(int) * row_ptrs_count);
                    (*row_ptrs)[row_ptrs_count - 1] = data_count;
                    inserted_row_ptr = 1;
                }
                data_count++;
                *data = (int *) realloc(*data, sizeof(int) * data_count);
                *col_offsets = (int *) realloc(*col_offsets, sizeof(int) * data_count);
                (*data)[data_count - 1] = mat[ele_ind];
                (*col_offsets)[data_count - 1] = j;
            }
        }
        if (inserted_row_ptr == 0) {
            row_ptrs_count++;
            *row_ptrs = (int *) realloc(*row_ptrs, sizeof(int) * row_ptrs_count);
            (*row_ptrs)[row_ptrs_count - 1] = data_count;
        }
    }
    row_ptrs_count++;
    *row_ptrs = (int *) realloc(*row_ptrs, sizeof(int) * row_ptrs_count);
    (*row_ptrs)[row_ptrs_count - 1] = data_count;
    *ret_row_ptrs_count = row_ptrs_count;
    *ret_data_count = data_count;
}

int main() {

    int r1, c1;

    // User input

    // printf("Enter dimensions of sparse matrix ");
    // scanf("%d %d", &r1, &c1);

    // printf("Enter input for matrix 1:\n");
    // int * mat = (int *) malloc(sizeof(int) * r1 * c1);
    // for (int i = 0; i < r1; i++) {
    //     for (int j = 0; j < c1; j++) {
    //         printf("Enter mat[%d][%d] ", i, j);
    //         scanf("%d", &mat[i * c1 + j]);
    //     }
    // }

    // printf("Enter input for vector:\n");
    // int * vector = (int *) malloc(sizeof(int) * c1);
    // for (int i = 0; i < c1; i++) {
    //     printf("Enter vector[%d] ", i);
    //     scanf("%d", &vector[i]);
    // }

    // Hardcoded input

    r1 = 4, c1 = 4;

    int mat[] = {1, 0, 3, 0, 0, 0, 0, 0, 0, 2, 4, 0, 7, 8, 0, 0};
    int vector[] = {1, 2, 3, 4};

    // End of input

    printf("Sparse Matrix:\n");
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c1; j++) {
            printf("%d ", mat[i * c1 + j]);
        }
        printf("\n");
    }

    printf("Vector:\n");
    for (int i = 0; i < c1; i++) {
        printf("%d ", vector[i]);
    }
    printf("\n");

    // Representing in CSR format

    int * mat_row_ptrs = (int *) malloc(sizeof(int)), mat_row_ptrs_count = 0;
    int * mat_col_offsets = (int *) malloc(sizeof(int)); // col_offsets_count is same as data_count
    int * mat_data = (int *) malloc(sizeof(int)), mat_data_count = 0;

    to_csr_format(mat, r1, c1, &mat_row_ptrs, &mat_col_offsets, &mat_data, &mat_row_ptrs_count, &mat_data_count);

    printf("CSR format:\n");
    printf("Row pointers: ");
    for (int i = 0; i < mat_row_ptrs_count; i++) {
        printf("%d ", mat_row_ptrs[i]);
    }
    printf("\n");
    printf("Column offsets: ");
    for (int i = 0; i < mat_data_count; i++) {
        printf("%d ", mat_col_offsets[i]);
    }
    printf("\n");
    printf("Data: ");
    for (int i = 0; i < mat_data_count; i++) {
        printf("%d ", mat_data[i]);
    }
    printf("\n");

    int * d_vector, * d_result, * d_row_ptrs, * d_col_offsets, * d_data, * result;

    hipMalloc(&d_vector, sizeof(int) * c1);
    hipMalloc(&d_result, sizeof(int) * r1);
    hipMalloc(&d_row_ptrs, sizeof(int) * mat_row_ptrs_count);
    hipMalloc(&d_col_offsets, sizeof(int) * mat_data_count);
    hipMalloc(&d_data, sizeof(int) * mat_data_count);

    hipMemcpy(d_vector, vector, sizeof(int) * c1, hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptrs, mat_row_ptrs, sizeof(int) * mat_row_ptrs_count, hipMemcpyHostToDevice);
    hipMemcpy(d_col_offsets, mat_col_offsets, sizeof(int) * mat_data_count, hipMemcpyHostToDevice);
    hipMemcpy(d_data, mat_data, sizeof(int) * mat_data_count, hipMemcpyHostToDevice);

    spmv<<<1, r1>>>(d_row_ptrs, d_col_offsets, d_data, d_vector, d_result, c1);

    result = (int *) malloc(sizeof(int) * r1);
    hipMemcpy(result, d_result, sizeof(int) * r1, hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < r1; i++) {
        printf("%d ", result[i]);
    }
    printf("\n");

    free(mat_row_ptrs);
    free(mat_col_offsets);
    free(mat_data);
    free(result);

    hipFree(d_vector);
    hipFree(d_result);
    hipFree(d_row_ptrs);
    hipFree(d_col_offsets);
    hipFree(d_data);

    return 0;
}