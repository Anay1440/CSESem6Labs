#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void mat_manipulation(int * mat, int * res_mat, int cols) {
    int curr_row = threadIdx.x;

    for (int i = 0; i < cols; i++) {
        int val = mat[curr_row * cols + i];
        for (int j = 0; j < curr_row; j++) {
            val *= mat[curr_row * cols + i];
        }
        res_mat[curr_row * cols + i] = val;
    }
}

int main() {

    int r1, c1;

    // User input

    // printf("Enter dimensions of sparse matrix ");
    // scanf("%d %d", &r1, &c1);

    // printf("Enter input for matrix 1:\n");
    // int * mat = (int *) malloc(sizeof(int) * r1 * c1);
    // for (int i = 0; i < r1; i++) {
    //     for (int j = 0; j < c1; j++) {
    //         printf("Enter mat[%d][%d] ", i, j);
    //         scanf("%d", &mat[i * c1 + j]);
    //     }
    // }

    // Hardcoded input

    r1 = 4, c1 = 4;

    int mat[] = {1, 2, 3, 4, 1, 2, 3, 4, 1, 2, 3, 4, 1, 2, 3, 4};

    // End of input

    printf("Original Matrix:\n");
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c1; j++) {
            printf("%d ", mat[i * c1 + j]);
        }
        printf("\n");
    }

    int * d_mat, *result, *d_result;

    hipMalloc(&d_mat, sizeof(int) * r1 * c1);
    hipMalloc(&d_result, sizeof(int) * r1 * c1);

    hipMemcpy(d_mat, mat, sizeof(int) * r1 * c1, hipMemcpyHostToDevice);

    mat_manipulation<<<1, r1>>>(d_mat, d_result, c1);

    result = (int *) malloc(sizeof(int) * r1 * c1);
    hipMemcpy(result, d_result, sizeof(int) * r1 * c1, hipMemcpyDeviceToHost);

    printf("Resultant Matrix:\n");
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c1; j++) {
            printf("%d ", result[i * c1 + j]);
        }
        printf("\n");
    }

    hipFree(d_mat);
    hipFree(d_result);
    free(result);

    return 0;
}