#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void conv(int *a, int *mask, int *out, int *width, int * mask_width) {
    int n = *mask_width / 2;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int res = 0, ind = 0;
    for (int j = i - n; j <= i + n; j++) {
        if (j < 0 || j >= *width)
            continue;
        res += a[j] * mask[ind++];
    }
    out[i] = res;
}

int main(void) {
    int n, m;

    printf("Enter n and m ");
    scanf("%d %d", &n, &m);

    // Host copies of vectors a, mask, out
    int * a = (int *) malloc(n * sizeof(int));
    int * mask = (int *) malloc(m * sizeof(int));
    int * out = (int *) malloc(n * sizeof(int));

    // Setup input values
    printf("Enter input array\n");
    for (int i = 0; i < n; i++) {
        printf("Enter a[%d] ", i);
        scanf("%d", &a[i]);
    }

    printf("Enter mask array\n");
    for (int i = 0; i < m; i++) {
        printf("Enter mask[%d] ", i);
        scanf("%d", &mask[i]);
    }

    int *d_a, *d_mask, *d_out, *d_width, *d_mask_width; // Device copies of vectors a, mask, out, width, mask_width
    int size = n * sizeof(int);
    int mask_size = m * sizeof(int);

    // Allocate space for device copies

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_mask, mask_size);
    hipMalloc((void **)&d_out, size);
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_mask_width, sizeof(int));

    // Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, mask_size, hipMemcpyHostToDevice);
    hipMemcpy(d_width, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_width, &m, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on GPU

    conv<<<1, n>>>(d_a, d_mask, d_out, d_width, d_mask_width);

    // Copy result back to host

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++) 
        printf("%d ", out[i]);

    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_mask);
    hipFree(d_out);
    hipFree(d_mask_width);
    return 0;

}