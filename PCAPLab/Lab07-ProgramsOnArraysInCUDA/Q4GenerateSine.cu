#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void conv(double *a, double *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = sin(a[i]);
}

int main(void) {
    int n;

    printf("Enter n ");
    scanf("%d", &n);

    // Host copies of vectors a, mask, out
    double * a = (double *) malloc(n * sizeof(double));
    double * out = (double *) malloc(n * sizeof(double));

    // Setup input values
    printf("Enter input array\n");
    for (int i = 0; i < n; i++) {
        printf("Enter a[%d] ", i);
        scanf("%lf", &a[i]);
    }

    double *d_a, *d_out; // Device copies
    int size = n * sizeof(double);

    // Allocate space for device copies

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_out, size);

    // Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // Launch kernel on GPU

    conv<<<1, n>>>(d_a, d_out);

    // Copy result back to host

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++) 
        printf("%.2lf ", out[i]);

    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_out);
    return 0;

}