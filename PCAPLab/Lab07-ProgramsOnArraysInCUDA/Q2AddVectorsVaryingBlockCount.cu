#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main(void) {
    int n;

    printf("Enter n ");
    scanf("%d", &n);

    int block_count = (n / 257) + 1;

    printf("Using %d blocks with 256 threads in each block\n", block_count);

    // Host copies of vectors a, b, c
    int * a = (int *) malloc(n * sizeof(int));
    int * b = (int *) malloc(n * sizeof(int));
    int * c = (int *) malloc(n * sizeof(int));

    // Setup input values
    for (int i = 0; i < n; i++) {
        // printf("Enter a[%d] and b[%d] ", i, i);
        // scanf("%d %d", &a[i], &b[i]);

        // Hardcoded inp
        a[i] = 1, b[i] = 2;
    }

    int *d_a, *d_b, *d_c; // Device copies of vectors
    int size = n * sizeof(int);

    // Allocate space for device copies

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU

    add<<<block_count, 256>>>(d_a, d_b, d_c);

    // Copy result back to host

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++) 
        printf("%d ", c[i]);

    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;

}