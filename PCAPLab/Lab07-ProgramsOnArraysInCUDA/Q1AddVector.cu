#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main(void) {
    int n;

    printf("Enter n ");
    scanf("%d", &n);

    // Host copies of vectors a, b, c
    int * a = (int *) malloc(n * sizeof(int));
    int * b = (int *) malloc(n * sizeof(int));
    int * c = (int *) malloc(n * sizeof(int));

    // Setup input values
    for (int i = 0; i < n; i++) {
        printf("Enter a[%d] and b[%d] ", i, i);
        scanf("%d %d", &a[i], &b[i]);
    }

    int *d_a, *d_b, *d_c; // Device copies of vectors a, b, c
    int size = n * sizeof(int);

    // Allocate space for device copies of a, b, c

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU

    // Part a
    add<<<1, n>>>(d_a, d_b, d_c);

    // Part b
    // add<<<n, 1>>>(d_a, d_b, d_c);

    // Copy result back to host

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++) 
        printf("%d ", c[i]);

    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;

}