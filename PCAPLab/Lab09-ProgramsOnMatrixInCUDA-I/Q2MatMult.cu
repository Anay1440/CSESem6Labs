#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void mat_mult_row_wise(int * a, int * b, int * c, int wa, int wb) {
    int row_a = threadIdx.x;
    for (int col_b = 0; col_b < wb; col_b++) {
        int sum = 0;
        for (int k = 0; k < wa; k++)
            sum += (a[row_a * wa + k] * b[k * wb + col_b]);

        c[row_a * wb + col_b] = sum;
    }
}

__global__ void mat_mult_col_wise(int * a, int * b, int * c, int wa, int ha) {
    int col_b = threadIdx.x;
    int wb = blockDim.x;
    for (int row_a = 0; row_a < ha; row_a++) {
        int sum = 0; 
        for (int k = 0; k < wa; k++)
            sum += (a[row_a * wa + k] * b[k * wb + col_b]);

        c[row_a * wb + col_b] = sum;
    }
}

__global__ void mat_mult_ele_wise(int * a, int *b, int * c, int wa) {
    int row_a = threadIdx.x;
    int col_b = blockIdx.x;
    int wb = gridDim.x;
    int sum = 0;
    for (int k = 0; k < wa; k++)
        sum += (a[row_a * wa + k] * b[k * wb + col_b]);

    c[row_a * wb + col_b] = sum;
}

int main() {

    int r1, r2, c1, c2, * d_a, * d_b, * d_c;

    // User Input

    // printf("Enter dimensions of first matrix ");
    // scanf("%d %d", &r1, &c1);
    // printf("Enter dimensions of second matrix ");
    // scanf("%d %d", &r2, &c2);

    // if (c1 != r2) {
    //     printf("Matrices with given dimensions cannot be multiplied\n");
    //     exit(1);
    // }

    // printf("Enter input for matrix 1:\n");
    // int * mat1 = (int *) malloc(sizeof(int) * r1 * c1);
    // for (int i = 0; i < r1; i++) {
    //     for (int j = 0; j < c1; j++) {
    //         printf("Enter mat1[%d][%d] ", i, j);
    //         scanf("%d", &mat1[i * c1 + j]);
    //     }
    // }

    // printf("Enter input for matrix 2:\n");
    // int * mat2 = (int *) malloc(sizeof(int) * r2 * c2);
    // for (int i = 0; i < r2; i++) {
    //     for (int j = 0; j < c2; j++) {
    //         printf("Enter mat2[%d][%d] ", i, j);
    //         scanf("%d", &mat2[i * c2 + j]);
    //     }
    // }

    // Hardcoded input

    r1 = 1, c1 = 3, r2 = 3, c2 = 4;

    int mat1[] = {3, 4, 2};
    int mat2[] = {13, 9, 7, 15, 8, 7, 4, 6, 6, 4, 0, 3};

    // End of input

    printf("Mat1:\n");
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c1; j++) {
            printf("%d ", mat1[i * c1 + j]);
        }
        printf("\n");
    }

    printf("Mat2:\n");
    for (int i = 0; i < r2; i++) {
        for (int j = 0; j < c2; j++) {
            printf("%d ", mat2[i * c2 + j]);
        }
        printf("\n");
    }

    int * res = (int *) malloc(sizeof(int) * r1 * c2);

    hipMalloc((void **)&d_a, r1 * c1 * sizeof(int));
    hipMalloc((void **)&d_b, r2 * c2 * sizeof(int));
    hipMalloc((void **)&d_c, r1 * c2 * sizeof(int));

    hipMemcpy(d_a, mat1, r1 * c1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, mat2, r2 * c2 * sizeof(int), hipMemcpyHostToDevice);

    // mat_mult_row_wise<<<1, r1>>>(d_a, d_b, d_c, c1, c2);

    // cudaMemcpy(res, d_c, r1 * c2 * sizeof(int), cudaMemcpyDeviceToHost);

    // printf("Result from rowwise:\n");

    // for (int i = 0; i < r1; i++) {
    //     for (int j = 0; j < c2; j++)
    //         printf("%d ", res[i * r1 + j]);
    //     printf("\n");
    // }

    // mat_mult_col_wise<<<1, c2>>>(d_a, d_b, d_c, c1, r1);

    // cudaMemcpy(res, d_c, r1 * c2 * sizeof(int), cudaMemcpyDeviceToHost);

    // printf("Result from columnwise:\n");

    // for (int i = 0; i < r1; i++) {
    //     for (int j = 0; j < c2; j++)
    //         printf("%d ", res[i * r1 + j]);
    //     printf("\n");
    // }

    mat_mult_ele_wise<<<c2, r1>>>(d_a, d_b, d_c, c1);

    hipMemcpy(res, d_c, r1 * c2 * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result from elementwise:\n");

    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c2; j++)
            printf("%d ", res[i * c1 + j]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}