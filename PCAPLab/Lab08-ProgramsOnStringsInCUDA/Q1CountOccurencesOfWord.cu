#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void count_occ_of_word(char * str, char * word, int * space_indices, unsigned int *d_count) {
    int i = threadIdx.x;
    int start, end;
    if (i == 0) {
        start = 0;
        end = space_indices[0] - 1;
    }
    else {
        start = space_indices[i - 1] + 1;
        end = space_indices[i] - 1;
    }

    int flag = 1, ind = 0;
    for (int j = start; j <= end; j++) {
        if (word[ind++] != str[j]) {
            flag = 0;
            break;
        }
    }
    if (flag)
        atomicAdd(d_count, 1);
}

int main() {
    char str[100], word[100];
    char * d_str, * d_word;
    int * d_space_indices;
    unsigned int count = 0, *d_count, result;

    printf("Enter a string ");
    scanf("%[^\n]s", str);
    fflush(stdin);
    printf("Enter the word ");
    scanf("%s", word);

    int i = 0, space_count = 0, *space_indices = (int *) malloc(sizeof(int));
    while (str[i] != '\0') {
        if (str[i] == ' ') {
            space_count++;
            space_indices = (int *) realloc(space_indices, space_count * sizeof(int));
            space_indices[space_count - 1] = i;
        }
        i++;
    }
    space_count++;
    space_indices = (int *) realloc(space_indices, space_count * sizeof(int));
    space_indices[space_count - 1] = strlen(str);

    hipMalloc((void **)&d_str, strlen(str) * sizeof(char));
    hipMalloc((void **)&d_word, strlen(word) * sizeof(char));
    hipMalloc((void **)&d_space_indices, space_count * sizeof(int));
    hipMalloc((void **)&d_count, sizeof(unsigned int));

    hipMemcpy(d_str, str, strlen(str) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, strlen(word) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_space_indices, space_indices, space_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    count_occ_of_word<<<1, space_count>>>(d_str, d_word, d_space_indices, d_count);

    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurences of word = %u\n", result);

    hipFree(d_str);
    hipFree(d_word);
    hipFree(d_space_indices);
    hipFree(d_count);
    return 0;
}