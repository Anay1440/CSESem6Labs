#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void pyramid_str(char * s, int l, char * rs) {
    int i = threadIdx.x;
    int start = (i * (2 * l + ((i - 1) * -1))) / 2; // Sum of AP
    for (int j = 0; j < l - i; j++) {
        rs[start + j] = s[j];
    }
}

int main() {
    char word[100];
    char * d_word, * d_res;
    int wordLen;

    printf("Enter a word ");
    scanf("%s", word);

    wordLen = strlen(word);

    int totalchars = (wordLen * (wordLen + 1)) / 2; // Sum of natural numbers till n

    char * res = (char *) malloc(sizeof(char) * (totalchars + 1));

    hipMalloc((void **)&d_word, wordLen * sizeof(char));
    hipMalloc((void **)&d_res, totalchars * sizeof(char));

    hipMemcpy(d_word, word, wordLen * sizeof(char), hipMemcpyHostToDevice);

    pyramid_str<<<1, wordLen>>>(d_word, wordLen, d_res);

    hipMemcpy(res, d_res, totalchars * sizeof(char), hipMemcpyDeviceToHost);

    res[totalchars] = '\0';

    printf("Final str: %s\n", res);

    hipFree(d_word);
    hipFree(d_res);
    return 0;
}