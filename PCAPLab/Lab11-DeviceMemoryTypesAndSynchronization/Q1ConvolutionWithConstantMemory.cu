#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__constant__ int mask_constant_mem[100];

__global__ void conv(int *a, int *out, int *width, int * mask_width) {
    int n = *mask_width / 2;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int res = 0, ind = 0;
    for (int j = i - n; j <= i + n; j++) {
        if (j < 0 || j >= *width)
            continue;
        res += a[j] * mask_constant_mem[ind++];
    }
    out[i] = res;
}

int main(void) {
    int n, m;

    printf("Enter n and m ");
    scanf("%d %d", &n, &m);

    // Host copies of vectors a, mask, out
    int * a = (int *) malloc(n * sizeof(int));
    int * mask = (int *) malloc(m * sizeof(int));
    int * out = (int *) malloc(n * sizeof(int));

    // Setup input values
    printf("Enter input array\n");
    for (int i = 0; i < n; i++) {
        printf("Enter a[%d] ", i);
        scanf("%d", &a[i]);
    }

    printf("Enter mask array\n");
    for (int i = 0; i < m; i++) {
        printf("Enter mask[%d] ", i);
        scanf("%d", &mask[i]);
    }

    int *d_a, *d_out, *d_width, *d_mask_width; // Device copies of vectors a, mask, out, width, mask_width
    int size = n * sizeof(int);
    int mask_size = m * sizeof(int);

    // Allocate space for device copies

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_out, size);
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_mask_width, sizeof(int));

    // Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_width, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_width, &m, sizeof(int), hipMemcpyHostToDevice);

    // Method 1 to copy to constant memory:

    hipMemcpyToSymbol(HIP_SYMBOL(mask_constant_mem), mask, mask_size, 0, hipMemcpyHostToDevice);

    // Method 2 to copy to constant memory:

    // int * d_mask;
    // cudaGetSymbolAddress((void **)&d_mask, mask_constant_mem);
    // cudaMemcpy(d_mask, mask, mask_size, cudaMemcpyHostToDevice);

    // Launch kernel on GPU

    conv<<<1, n>>>(d_a, d_out, d_width, d_mask_width);

    // Copy result back to host

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++) 
        printf("%d ", out[i]);

    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_mask_width);
    return 0;

}