#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define TILE_WIDTH 4
#define MAX_MASK_WIDTH 20

__constant__ int mask_constant_mem[100];

__global__ void conv(int *a, int *out, int *width, int * mask_width) {
    int n = *mask_width / 2;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int res = 0;
    
    __shared__ int shared_arr[TILE_WIDTH + MAX_MASK_WIDTH - 1];

    int halo_index_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    int halo_index_right = (blockIdx.x + 1) * blockDim.x + threadIdx.x;

    if (threadIdx.x >= blockDim.x - n)
        shared_arr[threadIdx.x - (blockDim.x - n)] = (halo_index_left >= 0) ? a[halo_index_left] : 0;

    if (threadIdx.x < n)
        shared_arr[threadIdx.x + blockDim.x + n] = (halo_index_right < *width) ? a[halo_index_right] : 0;

    shared_arr[n + threadIdx.x] = a[i];

    __syncthreads();

    for (int j = 0; j < *mask_width; j++)
        res += shared_arr[threadIdx.x + j] * mask_constant_mem[j];

    out[i] = res;
}

int main(void) {
    int n, m;

    printf("Enter n and m ");
    scanf("%d %d", &n, &m);

    // Host copies of vectors a, mask, out
    int * a = (int *) malloc(n * sizeof(int));
    int * mask = (int *) malloc(m * sizeof(int));
    int * out = (int *) malloc(n * sizeof(int));

    // Setup input values
    printf("Enter input array\n");
    for (int i = 0; i < n; i++) {
        printf("Enter a[%d] ", i);
        scanf("%d", &a[i]);
    }

    printf("Enter mask array\n");
    for (int i = 0; i < m; i++) {
        printf("Enter mask[%d] ", i);
        scanf("%d", &mask[i]);
    }

    int *d_a, *d_out, *d_width, *d_mask_width; // Device copies of vectors a, mask, out, width, mask_width
    int size = n * sizeof(int);
    int mask_size = m * sizeof(int);

    // Allocate space for device copies

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_out, size);
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_mask_width, sizeof(int));

    // Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_width, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_width, &m, sizeof(int), hipMemcpyHostToDevice);

    // Method 1 to copy to constant memory:

    hipMemcpyToSymbol(HIP_SYMBOL(mask_constant_mem), mask, mask_size, 0, hipMemcpyHostToDevice);

    // Launch kernel on GPU


    dim3 grid_conf(((n - 1) / TILE_WIDTH) + 1, 1, 1);
    dim3 block_conf(TILE_WIDTH, 1, 1);

    conv<<<grid_conf, block_conf>>>(d_a, d_out, d_width, d_mask_width);

    // Copy result back to host

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++) 
        printf("%d ", out[i]);

    printf("\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_mask_width);
    return 0;

}